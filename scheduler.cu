#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
extern "C" {
#include "header.h"
}

hipStream_t stream1, stream2;
int *t;
int *h_t;
int si = 0;
int mi;
int li;
int uindex = 0;
int idle[N][active_t];


__device__ uint get_smid(void) 
{
	uint ret;
	asm("mov.u32 %0, %smid;" : "=r"(ret) );
	return ret;
}

__device__ void* pnum(void* max)
{
	int limit = *((int *)max);
	printf("in task function\n");
	printf("%d",limit);
	int i, n, sum;
	for(n = 0; n < limit; n++)
	{
		i = 1;
		sum = 0;
		while(i < n){
			if(n % i == 0)
			{
				sum = sum + i;
			}
			i++;
		}
	}
	return NULL;
}

__device__ func f = pnum;

__global__ void scheduler(taskQueue_t *queues, complete_t *complete_d, int T, int *t)
{
	int sm = get_smid();
	if(threadIdx.x < active_t)
	{
		func g = NULL;
		while(*t < T)
		{
			if(queues[sm].f[threadIdx.x] != NULL)
			{
				queues[sm].f[threadIdx.x]((void *)&(queues[sm].arg[threadIdx.x]));
				queues[sm].f[threadIdx.x] = g;
				complete_d[sm].complete[threadIdx.x] += 1;
				atomicAdd(t, 1);
			}
		}
	}
}

// schedule func(arg) to be invoked on GPU in taskQueue[sm], return taskId
int taskAdd(void *(*func) (void *), void *arg, int sm)
{
	queues_h[sm].f[queues_h[sm].count] = func;
	queues_h[sm].arg[queues_h[sm].count] = *((int *)arg);
	int taskID = queues_h[sm].count * 10 + sm;	
	queues_h[sm].count += 1;
	return taskID;
}

void add_to_queue(int tasks[], int sort, int num_st, int num_mt, int num_lt, int taskIDs[][active_t], int sm, func h_f, int T)
{
	int i,j;
	int max;
	func g = NULL; 
	if(sort == 1)
	{
		if(sm == 0 | sm == 1)
		{
			j = si;
			max = num_st;
			for(i = 0; i < active_t; j++, i++)
			{       
				if(j < max)
				{       
					taskIDs[sm][i] = taskAdd(h_f, (void *)&tasks[j], sm);
					si++;
				}
				else    
				{       
					taskIDs[sm][i] = taskAdd(g, (void*)&tasks[j], sm);
				}
			}
		}
		else if(sm == 2 | sm == 3)
		{
			j = mi;
			max = num_st + num_mt;
			for(i = 0; i < active_t; j++, i++)
			{       
				if(j < max)
				{       
					taskIDs[sm][i] = taskAdd(h_f, (void *)&tasks[j], sm);
					mi++;
				}
				else    
				{       
					taskIDs[sm][i] = taskAdd(g, (void*)&tasks[j], sm);
				}
			}
		}
		else
		{
			j = li;
			max = T;
			for(i = 0; i < active_t; j++, i++)
			{       
				if(j < max)
				{       
					taskIDs[sm][i] = taskAdd(h_f, (void *)&tasks[j], sm);
					li++;
				}
				else    
				{       
					taskIDs[sm][i] = taskAdd(g, (void*)&tasks[j], sm);
				}
			}
		}
	}
	else if(sort == 0)
	{
		j = uindex;
		max = T;
		for(i = 0; i < active_t; j++, i++)
		{
			if(j < max)
			{
				if(tasks[j] == 100)
					si++;
				else if(tasks[j] == 400)
					mi++;
				else if(tasks[j] == 1000)
					li++;
				taskIDs[sm][i] = taskAdd(h_f, (void *)&tasks[j], sm);
				uindex++;
			}
			else
			{
				taskIDs[sm][i] = taskAdd(g, (void *)&tasks[j], sm);
			}
		}
	}
	for(i = 0; i < active_t; i++)
	{
		h_complete[sm].complete[i] = complete_h[sm].complete[i];
		h_queues[sm].arg[i] = queues_h[sm].arg[i];
		h_queues[sm].f[i] = queues_h[sm].f[i];
		h_queues[sm].count = queues_h[sm].count;
	}
}

void printoutput()
{
	printf("Done NXM matrix:\n");
	int i,j;
	for(i = 0; i < N; i++)
	{
		for(j = 0; j < active_t; j++)
		{
			if(j == 0)
				printf("%d:- ", i);
			printf("%10d ", h_complete[i].complete[j]);
		}
		printf("\n");
	}
	printf("\n\n\nCount NxM matrix:\n");
	for(i = 0; i < N; i++)
	{
		for(j = 0; j < active_t; j++)
		{
			if(j == 0)
				printf("%d:- ", i);
			printf("%10d ", idle[i][j]);
		}
		printf("\n");
	}
}

// check if task taskId is done, returns TRUE/FALSE
int taskDone(int taskId)
{
	int sm = taskId % 10;
	int index = taskId / 10;
	if(h_complete[sm].complete[index] == (dindex[sm] + 1))
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

void wrapper(int M, int T, int tasks[], int sort, int num_st, int num_mt, int num_lt)
{
	complete_t *complete_d;
	taskQueue_t *queues_d;

	int taskIDs[N][active_t];
	mi = num_st;
	li = num_st + num_mt;

	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);

	func h_f;

	hipMemcpyFromSymbol(&h_f, HIP_SYMBOL(f), sizeof(func));

	//hipMalloc
	hipMalloc((void **)&queues_d, N * sizeof(taskQueue_t));
	hipMalloc((void **)&complete_d, N * sizeof(complete_t));
	hipMalloc((void **)&t, sizeof(int));
	hipHostMalloc(&h_queues, N * sizeof(taskQueue_t));
	hipHostMalloc(&h_complete, N * sizeof(complete_t));
	hipHostMalloc(&h_t, sizeof(int));

	int i;
	for(i = 0; i < N; i++)
	{
		add_to_queue(tasks, sort, num_st, num_mt, num_lt, taskIDs, i, h_f, T);
	}

	//hipMemcpy
	*h_t = 0;
	hipMemcpy(t, (void *)h_t, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(queues_d, (void *)&queues_h, N * sizeof(taskQueue_t), hipMemcpyHostToDevice);
	hipMemcpy(complete_d, (void *)&complete_h, N * sizeof(complete_t), hipMemcpyHostToDevice);

	//kernel call
	scheduler<<<N, M, 0, stream1>>>(queues_d, complete_d, T, t);

	//while all tasks are not done
	while(1)
	{
		int i;
		//memcpyasync
		hipMemcpyAsync(h_complete, complete_d, N * sizeof(complete_t), hipMemcpyDeviceToHost, stream2);
		hipMemcpyAsync(h_t, t, sizeof(int), hipMemcpyDeviceToHost, stream2);
		for(i = 0; i < N; i++)
		{
			int j;
			int done = 1;
			for(j = 0; j < active_t; j++)
			{
				if((taskDone(taskIDs[i][j])) == 0)
				{
					done = 0;
				}
				else
				{
					idle[i][j] = idle[i][j] + 1;
				}	
			}
			int flag = 0;
			if(done)
			{
				if(sort)
				{
					if(i == 0 | i == 1)
					{
						if(si < num_st)
						{
							flag = 1;
						}
					}	
					if(i == 2 | i == 3)
					{
						if(mi < num_st + num_mt)
						{
							flag = 1;
						}
					}
					if(i == 4 | i == 5)
					{
						if(li < T)
						{
							flag = 1;
						}	
					}
				}
				else
				{
					flag = 1;
				}
				if(flag == 1)
				{
					queues_h[i].count = 0;
					dindex[i]++;
					add_to_queue(tasks, sort, num_st, num_mt, num_lt, taskIDs, i, h_f, T);
					hipMemcpyAsync((void *)&queues_d[i], (void *)&h_queues[i], sizeof(taskQueue_t), hipMemcpyHostToDevice, stream2);
				}
			}
		}
		if(*h_t == T & si == num_st & mi == (num_st+num_mt) & li == T)
		{
			printf("all tasks finished: %d tasks in total\n", *h_t);
			printoutput();
			break;
		}
	}
	//hipStreamDestroy(stream1);
	//hipStreamDestroy(stream2);
	//hipHostFree(&h_queues);
	//hipHostFree(&h_complete);
}
