/*
 * nvcc -arch=compute_20 -code="sm_20,compute_20" -o smid smid.cu
 * ./smid 20
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include <cuda_runtime.h>

/* E.D. Riedijk */

__device__ uint get_smid(void) {
  uint ret;
  asm("mov.u32 %0, %smid;" : "=r"(ret) );
  return ret;
}

__global__ void kern(int *sm){
  if (threadIdx.x==0)
    sm[blockIdx.x]=get_smid();
}

int main(int argc, char *argv[]){
  int N = atoi(argv[1]);
  int *sm, *sm_d;
  sm = (int *) malloc(N*sizeof(*sm));
  hipMalloc((void**)&sm_d,N*sizeof(*sm_d));
  kern<<<N,N>>>( sm_d);
  hipMemcpy(sm, sm_d, N*sizeof(int), hipMemcpyDeviceToHost);

  for (int i=0;i<N;i++)
    printf("%d %d\n",i,sm[i]);

  return 0;
}
